#include <assert.h>
#include <stdio.h>
#include <algorithm>
#include <stdlib.h>
#include<iostream>
#include "hip/hip_runtime.h"




#define NUM   (256*1024*1024)

#define THREADS_PER_BLOCK_X  384
#define THREADS_PER_BLOCK_Y  1
#define THREADS_PER_BLOCK_Z  1

#define PROTECT_BITS  (0xFFFF0000)

__global__ void
test_kernel( int* __restrict__ buf, int protectBits, int shrinkBits)
{

	int x = blockDim.x * blockIdx.x + threadIdx.x;

	int address;
	address = (x & protectBits) | (x & shrinkBits);

	buf[address] = x;
	//printf("address[%d] tid:%d \n ",address,x);
}


using namespace std;

int main() {

	int* hostA;

	int* deviceA;


	hipEvent_t start, stop;

	hipEventCreate (&start);
	hipEventCreate (&stop);

	float eventMs = 1.0f;


	hostA = (int*)malloc(NUM * sizeof(int));


	hipMalloc((void**)& deviceA, NUM * sizeof(int));
	hipMemcpy(deviceA, hostA, NUM * sizeof(int), hipMemcpyHostToDevice);

	
	test_kernel<<<dim3(1,1,1),dim3(1,1,1),0,0>>>( deviceA , 0x0, 0x0);

	for (int i = 16; i < 64 * 1024; i = i << 1) {

		hipEventRecord(start, 0);
		test_kernel<<<dim3(NUM/THREADS_PER_BLOCK_X, 1, 1),dim3(THREADS_PER_BLOCK_X, 1, 1),0,0>>>(deviceA,PROTECT_BITS,i - 1);

		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&eventMs, start, stop);

		printf("elapsed time:%f\n", eventMs);
		int bandwidth = (double)NUM * sizeof(int) / 1024 / 1024 / 1024 / (eventMs / 1000);
		printf("Shrink Size in Bytes[%ld], bandwidth %d (GB/S)\n", i*sizeof(int), bandwidth);

	}

	hipFree(deviceA);

	free(hostA);

	return 0;
}

